#include "hip/hip_runtime.h"
// ajee10x
/*
(MIN / MAX searches, dividing, multiplying) 
Operations for large array sizes in a serial form on the CPU and parallel on the GPU.
*/

#include "hip/hip_runtime.h" 
#include "" 
#include <iostream> 
#include <stdio.h>
#include <math.h>

void sum(float * a, float * b, float * c, int n);

//draw ، runs in parallel on a large number of threads
__global__ void sumKernel(float * a, float * b, float * c) {
  //Global Thread Index in Grid
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  //Sum the data corresponding to the given thread
  c[idx] = a[idx] + b[idx];
}

int main(int argc, char * argv[]) {
  const int n = 10000;
  float * a, * b, * c;
  a = new float[n];
  b = new float[n];
  c = new float[n];
  for (int i = 0; i < n; i++) {
    a[i] = rand() / (float) RAND_MAX - 0.5 f;
    b[i] = rand() / (float) RAND_MAX - 0.5 f;
  }
  sum(a, b, c, n);
  for (int i = 0; i < n; i++) {
    printf("c[%d]=%f\n", i, c[i]);
  }
  return 0;
}

void sum(float * a, float * b, float * c, int n) {
  //Total array size in bytes
  int numBytes = n * sizeof(float);

  //GPU pointer declaration
  float * aDev = NULL;
  float * bDev = NULL;
  float * cDev = NULL;

  float gpuTime = 0.0 f;
  hipEvent_t start, stop;

  //GPU memory allocation
  hipMalloc((void ** ) & aDev, numBytes);
  hipMalloc((void ** ) & bDev, numBytes);
  hipMalloc((void ** ) & cDev, numBytes);
  //Determining the number of blocks in a grid and threads in a block
  dim3 threads = dim3(512);
  dim3 blocks = dim3(n / threads.x);
  //Copying input data from CPU memory to GPU memory
  hipMemcpy(aDev, a, numBytes, hipMemcpyHostToDevice);
  hipMemcpy(bDev, b, numBytes, hipMemcpyHostToDevice);

  hipEventCreate( & start);
  hipEventCreate( & stop);
  hipEventRecord(start, 0);

  //Running the summation kernel on the GPU with a given configuration of blocks and threads
  sumKernel << < blocks, threads >> > (aDev, bDev, cDev);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime( & gpuTime, start, stop);
  hipEventRecord(start, 0);
  printf("time in miliseconds is %.4f\n", gpuTime);

  //Copy result from GPU to CPU
  hipMemcpy(c, cDev, numBytes, hipMemcpyDeviceToHost);
  //Freeing memory on the GPU
  hipFree(aDev);
  hipFree(bDev);
  hipFree(cDev);

  hipEventDestroy(start);
  hipEventDestroy(stop);
}
